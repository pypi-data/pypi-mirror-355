#include "hip/hip_runtime.h"
#include "../util/cuda_error.h"
#include "../util/inline_ops.cuh"
#include <hipblas.h>

#include "flashrnn.h"
#include "flashrnn_pointwise.cuh"

#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

#ifndef FLASHRNN_NUM_GATES_T
#define FLASHRNN_NUM_GATES_R 4
#define FLASHRNN_NUM_GATES_W 4
#define FLASHRNN_NUM_GATES_I 4
#define FLASHRNN_NUM_GATES_T 4
#define FLASHRNN_GRADIENT_RECURRENT_CLIPVAL 0.
#define FLASHRNN_GRADIENT_RECURRENT_CLIPVAL_VALID false
#define FLASHRNN_DTYPE_G float
#define FLASHRNN_DTYPE_B float
#define FLASHRNN_DTYPE_S float
#endif

static_assert(FLASHRNN_NUM_GATES_T == 4, "Total gates must be 4");
static_assert(FLASHRNN_NUM_GATES_I == 4, "Interacting gates must be 4");
static_assert(FLASHRNN_NUM_GATES_W == 4, "Input-based gates must be 4");
static_assert(FLASHRNN_NUM_GATES_R == 4, "Recurrent gates must be 4");

namespace flashrnn {

template <bool Training>
__global__ void FLASHRNNPointwiseForward(
    const int batch_dim, const int hidden_dim, const int num_heads,
    const FLASHRNN_DTYPE_G *Wx, // Precomputed (Wx) vector
    const FLASHRNN_DTYPE_G *Ry, // Precomputed (Ry) vector
    const FLASHRNN_DTYPE_B *b,  // Bias for gates
    const FLASHRNN_DTYPE_S *s,  // Input  state
    const uint s_stride,
    FLASHRNN_DTYPE_S *s_out, // Output recurrent state
    const uint s_out_stride,
    FLASHRNN_DTYPE_G *g_r_out, // Output vector v (Wx + Ry + b) (only
                               // used if autoraining==true)
    FLASHRNN_DTYPE_G *g_i_out) {

  // We're in column-major order here, so increase x => increase row.
  const int row = blockDim.x * blockIdx.x + threadIdx.x; // hidden
  const int col = blockDim.y * blockIdx.y + threadIdx.y; // batch
  const int head_dim = hidden_dim / num_heads;
  const int head_idx = (blockDim.z * blockIdx.z + threadIdx.z) * head_dim;

  if (row >= head_dim || col >= batch_dim)
    return;

  // Base index into the Wx and Ry matrices.
  const int weight_idx = col * (hidden_dim * FLASHRNN_NUM_GATES_R) + row +
                         FLASHRNN_NUM_GATES_R * head_idx;

  // Base index into the output matrix. autohis is different from `weight_idx`
  // because the number of rows are different between the two sets of matrices.
  const int output_idx = col * hidden_dim + row + head_idx;

  const int i_idx = weight_idx + 0. * head_dim;
  const int f_idx = weight_idx + 1. * head_dim;
  const int z_idx = weight_idx + 2. * head_dim;
  const int o_idx = weight_idx + 3. * head_dim;

  const auto c_cur = type2float(s[output_idx + 1 * s_stride]);
  auto n_cur = type2float(s[output_idx + 2 * s_stride]);
  const auto m_cur = type2float(s[output_idx + 3 * s_stride]);
  const auto iraw = add_g(
      type2float(Wx[i_idx]),
      add_g(
          type2float(Ry[i_idx]),
          type2float(b[row + FLASHRNN_NUM_GATES_T * head_idx + 0 * head_dim])));
  const auto fraw = add_g(
      type2float(Wx[f_idx]),
      add_g(
          type2float(Ry[f_idx]),
          type2float(b[row + FLASHRNN_NUM_GATES_T * head_idx + 1 * head_dim])));
  const auto logfplusm = add_g(logsigmoid_g(fraw), m_cur);
  const auto zval = tanh_g(add_g(
      type2float(Wx[z_idx]),
      add_g(type2float(Ry[z_idx]),
            type2float(
                b[row + FLASHRNN_NUM_GATES_T * head_idx + 2 * head_dim]))));
  const float one = 1.;
  float m_new;
  if (eq_zero_g(n_cur)) {
    m_new = iraw;
#ifdef FLASHRNN_STABILIZATION_EPSILON_FIRST
    n_cur =
        add_g(n_cur, float2type<auto>(FLASHRNN_STABILIZATION_EPSILON_FIRST));
#endif
  } else {
    m_new = max_g(iraw, logfplusm);
  }
#ifdef FLASHRNN_STABILIZATION_EPSILON
  n_cur = add_g(n_cur, float2type<auto>(FLASHRNN_STABILIZATION_EPSILON));
#endif
  const auto ogate = sigmoid_g(add_g(
      type2float(Wx[o_idx]),
      add_g(type2float(Ry[o_idx]),
            type2float(
                b[row + FLASHRNN_NUM_GATES_T * head_idx + 3 * head_dim]))));

  const auto igate = min_g(one, exp_g(sub_g(iraw, m_new)));
  const auto fgate = min_g(one, exp_g(sub_g(logfplusm, m_new)));
  // Compile-time constant branch should be eliminated by compiler so we have
  // straight-through code.
  if (Training) {
    g_r_out[i_idx] = float2type<FLASHRNN_DTYPE_G>(igate);
    g_r_out[f_idx] = float2type<FLASHRNN_DTYPE_G>(fraw);
    g_r_out[z_idx] = float2type<FLASHRNN_DTYPE_G>(zval);
    g_r_out[o_idx] = float2type<FLASHRNN_DTYPE_G>(ogate);
  }

  const auto c_new = add_g(mul_g(fgate, c_cur), mul_g(igate, zval));
  // n >= 1 by construction for non-initial state
  float n_new = max_g(add_g(mul_g(fgate, n_cur), igate), one);
#ifdef FLASHRNN_STABILIZATION_EPSILON_NEW
  n_new = add_g(n_new, float2type<auto>(FLASHRNN_STABILIZATION_EPSILON_NEW));
#endif
  auto y_new = mul_g(ogate, div_g(c_new, n_new));

#if FLASHRNN_FORWARD_CLIPVAL_VALID
  y_new = clip_val_g(y_new, neg_g((float)FLASHRNN_FORWARD_CLIPVAL),
                     (float)FLASHRNN_FORWARD_CLIPVAL);
#endif

  s_out[output_idx + 0 * s_out_stride] = float2type<FLASHRNN_DTYPE_S>(y_new);
  s_out[output_idx + 1 * s_out_stride] = float2type<FLASHRNN_DTYPE_S>(c_new);
  s_out[output_idx + 2 * s_out_stride] = float2type<FLASHRNN_DTYPE_S>(n_new);
  s_out[output_idx + 3 * s_out_stride] = float2type<FLASHRNN_DTYPE_S>(m_new);
}

__global__ void FLASHRNNPointwiseBackward(
    const int batch_dim, const int hidden_dim, const int num_heads,
    const FLASHRNN_DTYPE_S *s, const uint s_stride, const FLASHRNN_DTYPE_G *g_r,
    const FLASHRNN_DTYPE_G *g_i,
    const FLASHRNN_DTYPE_B *b, // Bias for gates
    const FLASHRNN_DTYPE_S *s_new, const uint s_new_stride,
    const FLASHRNN_DTYPE_S *ds_new, const uint ds_new_stride,
    FLASHRNN_DTYPE_S *ds_inout, const uint ds_inout_stride,
    FLASHRNN_DTYPE_G *dg_r_out, FLASHRNN_DTYPE_G *dg_i_out,
    FLASHRNN_DTYPE_G *dg_b_out) {
  const int row = blockDim.x * blockIdx.x + threadIdx.x; // hidden
  const int col = blockDim.y * blockIdx.y + threadIdx.y; // batch
  const int head_dim = hidden_dim / num_heads;
  const int head_idx = (blockDim.z * blockIdx.z + threadIdx.z) * head_dim;

  if (row >= head_dim || col >= batch_dim)
    return;

  const int base_idx = col * hidden_dim + row + head_idx;
  auto dy_recurrent = type2float(ds_inout[base_idx + 0 * ds_inout_stride]);

#if (FLASHRNN_GRADIENT_RECURRENT_CLIPVAL_VALID)
  dy_recurrent = clip_val_g(dy_recurrent,
                            neg_g((float)FLASHRNN_GRADIENT_RECURRENT_CLIPVAL),
                            (float)FLASHRNN_GRADIENT_RECURRENT_CLIPVAL);
#endif
  const auto dy_total =
      add_g(type2float(ds_new[base_idx + 0 * ds_new_stride]), dy_recurrent);
  auto dc_total = add_g(type2float(ds_new[base_idx + 1 * ds_new_stride]),
                        type2float(ds_inout[base_idx + 1 * ds_inout_stride]));
  auto dn_total = add_g(type2float(ds_new[base_idx + 2 * ds_new_stride]),
                        type2float(ds_inout[base_idx + 2 * ds_inout_stride]));

  const int stride4_base_idx = col * (hidden_dim * FLASHRNN_NUM_GATES_R) + row +
                               FLASHRNN_NUM_GATES_R * head_idx;
  const int i_idx = stride4_base_idx + 0 * head_dim;
  const int f_idx = stride4_base_idx + 1 * head_dim;
  const int z_idx = stride4_base_idx + 2 * head_dim;
  const int o_idx = stride4_base_idx + 3 * head_dim;

  const auto igate = type2float(g_r[i_idx]);
  const auto fraw = type2float(g_r[f_idx]);
  const auto zval = type2float(g_r[z_idx]);
  const auto ogate = type2float(g_r[o_idx]);
  const auto c_cur = type2float(s[base_idx + 1 * s_stride]);
  const auto n_cur = type2float(s[base_idx + 2 * s_stride]);
  const auto m_cur = type2float(s[base_idx + 3 * s_stride]);
  const float zero = 0.;
  const float one = 1.;
  const auto y_new = type2float(s_new[base_idx + 0 * s_new_stride]);
  const auto n_new = type2float(s_new[base_idx + 2 * s_new_stride]);
  const auto m_new = type2float(s_new[base_idx + 3 * s_new_stride]);

  const auto logfplusm = add_g(logsigmoid_g(fraw), m_cur);
  const auto fgate = min_g(one, exp_g(sub_g(logfplusm, m_new)));
  const auto fsig = min_g(one, sigmoid_g(fraw));
  const auto dy_inter = div_g(dy_total, n_new);
  dc_total = add_g(dc_total, mul_g(dy_inter, ogate));
  dn_total = sub_g(dn_total, mul_g(dy_inter, y_new));

  const auto dg_i = mul_g(igate, add_g(mul_g(zval, dc_total), dn_total));
  const auto dg_f =
      mul_g(mul_g(fgate, add_g(mul_g(dc_total, c_cur), mul_g(dn_total, n_cur))),
            sub_g(one, fsig));
  const auto dg_z = mul_g(mul_g(dc_total, igate), d_tanh_g(zval));
  const auto dg_o = mul_g(sub_g(one, ogate), mul_g(y_new, dy_total));

  const auto dc_i = mul_g(fgate, dc_total);
  const auto dn_i = mul_g(fgate, dn_total);

  ds_inout[base_idx + 0 * ds_inout_stride] = float2type<FLASHRNN_DTYPE_S>(zero);
  ds_inout[base_idx + 1 * ds_inout_stride] = float2type<FLASHRNN_DTYPE_S>(dc_i);
  ds_inout[base_idx + 2 * ds_inout_stride] = float2type<FLASHRNN_DTYPE_S>(dn_i);
  ds_inout[base_idx + 3 * ds_inout_stride] = float2type<FLASHRNN_DTYPE_S>(zero);

  dg_r_out[i_idx] = float2type<FLASHRNN_DTYPE_G>(dg_i);
  dg_r_out[f_idx] = float2type<FLASHRNN_DTYPE_G>(dg_f);
  dg_r_out[z_idx] = float2type<FLASHRNN_DTYPE_G>(dg_z);
  dg_r_out[o_idx] = float2type<FLASHRNN_DTYPE_G>(dg_o);
}

FLASHRNN_POST_DEFINITIONS

} // namespace flashrnn
